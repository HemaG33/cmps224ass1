#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

    // TODO







}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

