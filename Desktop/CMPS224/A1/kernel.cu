#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

    int i = blockDim.a*blockIdx.a + threadIdx.a;
	if(i < N) {
	    double aval = a[i];
        double bval = b[i];
        c[i] = (aval > bval)?aval:bval;
	}








}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

	float *a_d, *b_d, *c_d;
	hipMalloc((void**) &a_d, M*sizeof(float));
	hipMalloc((void**) &b_d, M*sizeof(float));
	hipMalloc((void**) &c_d, M*sizeof(float));




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

	hipMemcpy(a_d, x, M*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, y, M*sizeof(float), hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

	vecMax_kernel(




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

	hipMemcpy(c, c_d, M*sizeof(float), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

